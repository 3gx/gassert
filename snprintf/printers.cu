
#include <hip/hip_runtime.h>
#include <cstdio>

#ifndef __CUDACC__
#define __host__
#define __device__
#define __forceinline__
#endif

namespace printer {
enum {
  PRINT_F_MINUS          = (1 << 0),
  PRINT_F_PLUS           = (1 << 1),
  PRINT_F_SPACE          = (1 << 2),
  PRINT_F_NUM            = (1 << 3),
  PRINT_F_ZERO           = (1 << 4),
  PRINT_F_QUOTE          = (1 << 5),
  PRINT_F_UP             = (1 << 6),
  PRINT_F_UNSIGNED       = (1 << 7),
  PRINT_F_TYPE_G         = (1 << 8),
  PRINT_F_TYPE_E         = (1 << 9),
  MAX_CONVERT_LENGTH     = 43,
  OVERFLOW               = (1<<16),
  NaN                    = (1<<16)+1
};


__host__ __device__ __forceinline__
static bool ISNAN(double x) {return x != x; }

__host__ __device__ __forceinline__
static bool  ISINF(double x) { return x != 0.0 && x + x == x; }

__host__ __device__ __forceinline__
static void outchar(char *str, int &len, size_t &size, char ch)
{
  do
  {
    if (len + 1 < size)
      str[len] = ch;
    (len)++;
  } while (/* CONSTCOND */ 0);
}
__host__ __device__
static void printsep(char *str, int &len, size_t size)
{
		outchar(str, len, size, ',');
}


__host__ __device__
static int getexponent(double value)
{
	double tmp = (value >= 0.0) ? value : -value;
	int exponent = 0;

	/*
	 * We check for 99 > exponent > -99 in order to work around possible
	 * endless loops which could happen (at least) in the second loop (at
	 * least) if we're called with an infinite value.  However, we checked
	 * for infinity before calling this function using our ISINF() macro, so
	 * this might be somewhat paranoid.
	 */
	while (tmp < 1.0 && tmp > 0.0 && --exponent > -99)
		tmp *= 10;
	while (tmp >= 10.0 && ++exponent < 99)
		tmp /= 10;

	return exponent;
}

__host__ __device__
static double mypow10(int exponent)
{
	double result = 1;

	while (exponent > 0) {
		result *= 10;
		exponent--;
	}
	while (exponent < 0) {
		result /= 10;
		exponent++;
	}
	return result;
}

#define UINTMAX_MAX ((uint64_t)(-1))

__host__ __device__
static uint64_t cast(double value)
{
	uint64_t result;

	/*
	 * We check for ">=" and not for ">" because if UINTMAX_MAX cannot be
	 * represented exactly as an LDOUBLE value (but is less than LDBL_MAX),
	 * it may be increased to the nearest higher representable value for the
	 * comparison (cf. C99: 6.3.1.4, 2).  It might then equal the LDOUBLE
	 * value although converting the latter to UINTMAX_T would overflow.
	 */
	if (value >= UINTMAX_MAX)
		return UINTMAX_MAX;

	result = value;
	/*
	 * At least on NetBSD/sparc64 3.0.2 and 4.99.30, casting long double to
	 * an integer type converts e.g. 1.9 to 2 instead of 1 (which violates
	 * the standard).  Sigh.
	 */
	return (result <= value) ? result : result - 1;
}

__host__ __device__
static uint64_t myround(double value)
{
	double intpart = cast(value);

	return ((value -= intpart) < 0.5) ? intpart : intpart + 1;
}

__host__ __device__
static int getnumsep(int digits)
{
	int separators = (digits - ((digits % 3 == 0) ? 1 : 0)) / 3;
	return separators;
}


__host__ __device__
static int convert(uint64_t value, char *buf, size_t size, int base, int caps)
{
	const char *digits = caps ? "0123456789ABCDEF" : "0123456789abcdef";
	size_t pos = 0;

	/* We return an unterminated buffer with the digits in reverse order. */
	do {
		buf[pos++] = digits[value % base];
		value /= base;
	} while (value != 0 && pos < size);

	return (int)pos;
}
__host__ __device__ 
static int fmtflt(char *str, size_t size, double fvalue,
                   int width = 0, int precision = -1, int flags = 0)
{
  int len = 0;
	double ufvalue;
	uint64_t intpart;
	uint64_t fracpart;
	uint64_t mask;
	const char *infnan = NULL;
	char iconvert[MAX_CONVERT_LENGTH];
	char fconvert[MAX_CONVERT_LENGTH];
	char econvert[4];	/* "e-12" (without nul-termination). */
	char esign = 0;
	char sign = 0;
  
	int leadfraczeros = 0;
	int exponent = 0;
	int emitpoint = 0;
	int omitzeros = 0;
	int omitcount = 0;
	int padlen = 0;
	int epos = 0;
	int fpos = 0;
	int ipos = 0;
	int separators = (flags & PRINT_F_QUOTE);
	int estyle = (flags & PRINT_F_TYPE_E);

	/*
	 * AIX' man page says the default is 0, but C99 and at least Solaris'
	 * and NetBSD's man pages say the default is 6, and sprintf(3) on AIX
	 * defaults to 6.
	 */
	if (precision == -1)
		precision = 6;

	if (fvalue < 0.0)
		sign = '-';
	else if (flags & PRINT_F_PLUS)	/* Do a sign. */
		sign = '+';
	else if (flags & PRINT_F_SPACE)
		sign = ' ';

	if (ISNAN(fvalue))
		infnan = (flags & PRINT_F_UP) ? "NAN" : "nan";
	else if (ISINF(fvalue))
		infnan = (flags & PRINT_F_UP) ? "INF" : "inf";

	if (infnan != NULL) {
#if 0
		if (sign != 0)
			iconvert[ipos++] = sign;
		while (*infnan != '\0')
			iconvert[ipos++] = *infnan++;
		fmtstr(str, len, size, iconvert, width, ipos, flags);
#endif
		return len; /* NaN */
	}

	/* "%e" (or "%E") or "%g" (or "%G") conversion. */
	if (flags & PRINT_F_TYPE_E || flags & PRINT_F_TYPE_G) {
		if (flags & PRINT_F_TYPE_G) {
			/*
			 * For "%g" (and "%G") conversions, the precision
			 * specifies the number of significant digits, which
			 * includes the digits in the integer part.  The
			 * conversion will or will not be using "e-style" (like
			 * "%e" or "%E" conversions) depending on the precision
			 * and on the exponent.  However, the exponent can be
			 * affected by rounding the converted value, so we'll
			 * leave this decision for later.  Until then, we'll
			 * assume that we're going to do an "e-style" conversion
			 * (in order to get the exponent calculated).  For
			 * "e-style", the precision must be decremented by one.
			 */
			precision--;
			/*
			 * For "%g" (and "%G") conversions, trailing zeros are
			 * removed from the fractional portion of the result
			 * unless the "#" flag was specified.
			 */
			if (!(flags & PRINT_F_NUM))
				omitzeros = 1;
		}
		exponent = getexponent(fvalue);
		estyle = 1;
	}

again:
	/*
	 * Sorry, we only support 9, 19, or 38 digits (that is, the number of
	 * digits of the 32-bit, the 64-bit, or the 128-bit UINTMAX_MAX value
	 * minus one) past the decimal point due to our conversion method.
	 */
	switch (sizeof(uint64_t)) {
	case 16:
		if (precision > 38)
			precision = 38;
		break;
	case 8:
		if (precision > 19)
			precision = 19;
		break;
	default:
		if (precision > 9)
			precision = 9;
		break;
	}

	ufvalue = (fvalue >= 0.0) ? fvalue : -fvalue;
	if (estyle)	/* We want exactly one integer digit. */
		ufvalue /= mypow10(exponent);

	if ((intpart = cast(ufvalue)) == UINTMAX_MAX) {
    return 0; /* *overflow = 1 */
	}

	/*
	 * Factor of ten with the number of digits needed for the fractional
	 * part.  For example, if the precision is 3, the mask will be 1000.
	 */
	mask = mypow10(precision);
	/*
	 * We "cheat" by converting the fractional part to integer by
	 * multiplying by a factor of ten.
	 */
	if ((fracpart = myround(mask * (ufvalue - intpart))) >= mask) {
		/*
		 * For example, ufvalue = 2.99962, intpart = 2, and mask = 1000
		 * (because precision = 3).  Now, myround(1000 * 0.99962) will
		 * return 1000.  So, the integer part must be incremented by one
		 * and the fractional part must be set to zero.
		 */
		intpart++;
		fracpart = 0;
		if (estyle && intpart == 10) {
			/*
			 * The value was rounded up to ten, but we only want one
			 * integer digit if using "e-style".  So, the integer
			 * part must be set to one and the exponent must be
			 * incremented by one.
			 */
			intpart = 1;
			exponent++;
		}
	}

	/*
	 * Now that we know the real exponent, we can check whether or not to
	 * use "e-style" for "%g" (and "%G") conversions.  If we don't need
	 * "e-style", the precision must be adjusted and the integer and
	 * fractional parts must be recalculated from the original value.
	 *
	 * C99 says: "Let P equal the precision if nonzero, 6 if the precision
	 * is omitted, or 1 if the precision is zero.  Then, if a conversion
	 * with style `E' would have an exponent of X:
	 *
	 * - if P > X >= -4, the conversion is with style `f' (or `F') and
	 *   precision P - (X + 1).
	 *
	 * - otherwise, the conversion is with style `e' (or `E') and precision
	 *   P - 1." (7.19.6.1, 8)
	 *
	 * Note that we had decremented the precision by one.
	 */
	if (flags & PRINT_F_TYPE_G && estyle &&
	    precision + 1 > exponent && exponent >= -4) {
		precision -= exponent;
		estyle = 0;
		goto again;
	}

	if (estyle) {
		if (exponent < 0) {
			exponent = -exponent;
			esign = '-';
		} else
			esign = '+';

		/*
		 * Convert the exponent.  The sizeof(econvert) is 4.  So, the
		 * econvert buffer can hold e.g. "e+99" and "e-99".  We don't
		 * support an exponent which contains more than two digits.
		 * Therefore, the following stores are safe.
		 */
		epos = convert(exponent, econvert, 2, 10, 0);
		/*
		 * C99 says: "The exponent always contains at least two digits,
		 * and only as many more digits as necessary to represent the
		 * exponent." (7.19.6.1, 8)
		 */
		if (epos == 1)
			econvert[epos++] = '0';
		econvert[epos++] = esign;
		econvert[epos++] = (flags & PRINT_F_UP) ? 'E' : 'e';
	}

	/* Convert the integer part and the fractional part. */
	ipos = convert(intpart, iconvert, sizeof(iconvert), 10, 0);
	if (fracpart != 0)	/* convert() would return 1 if fracpart == 0. */
		fpos = convert(fracpart, fconvert, sizeof(fconvert), 10, 0);

	leadfraczeros = precision - fpos;

	if (omitzeros) {
		if (fpos > 0)	/* Omit trailing fractional part zeros. */
			while (omitcount < fpos && fconvert[omitcount] == '0')
				omitcount++;
		else {	/* The fractional part is zero, omit it completely. */
			omitcount = precision;
			leadfraczeros = 0;
		}
		precision -= omitcount;
	}

	/*
	 * Print a decimal point if either the fractional part is non-zero
	 * and/or the "#" flag was specified.
	 */
	if (precision > 0 || flags & PRINT_F_NUM)
		emitpoint = 1;
	if (separators)	/* Get the number of group separators we'll print. */
		separators = getnumsep(ipos);

	padlen = width                  /* Minimum field width. */
	    - ipos                      /* Number of integer digits. */
	    - epos                      /* Number of exponent characters. */
	    - precision                 /* Number of fractional digits. */
	    - separators                /* Number of group separators. */
	    - (emitpoint ? 1 : 0)       /* Will we print a decimal point? */
	    - ((sign != 0) ? 1 : 0);    /* Will we print a sign character? */

	if (padlen < 0)
		padlen = 0;

	/*
	 * C99 says: "If the `0' and `-' flags both appear, the `0' flag is
	 * ignored." (7.19.6.1, 6)
	 */
	if (flags & PRINT_F_MINUS)	/* Left justifty. */
		padlen = -padlen;
	else if (flags & PRINT_F_ZERO && padlen > 0) {
		if (sign != 0) {	/* Sign. */
			outchar(str, len, size, sign);
			sign = 0;
		}
		while (padlen > 0) {	/* Leading zeros. */
			outchar(str, len, size, '0');
			padlen--;
		}
	}
	while (padlen > 0) {	/* Leading spaces. */
		outchar(str, len, size, ' ');
		padlen--;
	}
	if (sign != 0)	/* Sign. */
		outchar(str, len, size, sign);
	while (ipos > 0) {	/* Integer part. */
		ipos--;
		outchar(str, len, size, iconvert[ipos]);
		if (separators > 0 && ipos > 0 && ipos % 3 == 0)
			printsep(str, len, size);
	}
	if (emitpoint) {	/* Decimal point. */
			outchar(str, len, size, '.');
	}
	while (leadfraczeros > 0) {	/* Leading fractional part zeros. */
		outchar(str, len, size, '0');
		leadfraczeros--;
	}
	while (fpos > omitcount) {	/* The remaining fractional part. */
		fpos--;
		outchar(str, len, size, fconvert[fpos]);
	}
	while (epos > 0) {	/* Exponent. */
		epos--;
		outchar(str, len, size, econvert[epos]);
	}
	while (padlen < 0) {	/* Trailing spaces. */
		outchar(str, len, size, ' ');
		padlen++;
	}

  return len;
} // void fmtflt(..)

} // namespace printer

int main(int argc, char * argv[])
{
  char str[256];
  printer::fmtflt(str, 256,  123.0f*argc);
  printf("%s \n", str);

}
